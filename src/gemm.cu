#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <iostream>

template <typename T>
void gemm_cpu(const size_t m, const size_t n, const size_t k, const T alpha,
              const T *A, const T *B, const T beta, T *C) {
  for (size_t im = 0; im < m; ++im) {
    for (size_t in = 0; in < n; ++in) {
      T value = 0;
      for (size_t ik = 0; ik < k; ++ik) {
        value += A[im * k + ik] * B[ik * n + in];
      }
      C[im * n + in] = alpha * value + beta;
    }
  }
}

template void gemm_cpu(const size_t m, const size_t n, const size_t k,
                       const float alpha, const float *A, const float *B,
                       const float beta, float *C);

template <typename T, int BLOCK_SIZE>
__global__ void gemm_kernel(const size_t m, const size_t n, const size_t k,
                            const T alpha, const T *A, const T *B, const T beta,
                            T *C) {
  __shared__ T shared_A[BLOCK_SIZE][BLOCK_SIZE]; // m x k
  __shared__ T shared_B[BLOCK_SIZE][BLOCK_SIZE]; // k x n

  const int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  const int idy = blockIdx.y * BLOCK_SIZE + threadIdx.y;

  T value = 0;

  int kb;
  for (kb = 0; kb < k / BLOCK_SIZE; ++kb) {
    int k_base = kb * BLOCK_SIZE;
    shared_A[threadIdx.y][threadIdx.x] = A[idy * k + (k_base + threadIdx.x)];
    shared_B[threadIdx.y][threadIdx.x] = B[(k_base + threadIdx.y) * n + idx];
    __syncthreads();

#pragma unroll
    for (int i = 0; i < BLOCK_SIZE; ++i) {
      value += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];
    }
    __syncthreads();
  }

  for (size_t i = kb * BLOCK_SIZE; i < k; ++i) {
    value += A[idy * k + i] * B[i * n + idx];
  }
  if (idx < n && idy < m) {
    C[idy * n + idx] = alpha * value + beta;
  }
}

template <typename T>
void gemm_cuda(const size_t m, const size_t n, const size_t k, const T alpha,
               const T *A, const T *B, const T beta, T *C) {
  T *d_A, *d_B, *d_C;

  CUDA_CHECK(hipMalloc((T **)&d_A, m * k * sizeof(T)));
  CUDA_CHECK(hipMalloc((T **)&d_B, k * n * sizeof(T)));
  CUDA_CHECK(hipMalloc((T **)&d_C, m * n * sizeof(T)));

  CUDA_CHECK(hipMemcpy(d_A, A, m * k * sizeof(T), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, B, k * n * sizeof(T), hipMemcpyHostToDevice));

  const int BLOCK_SIZE = 16;

  const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 grid(CEIL_DIV(n, BLOCK_SIZE), CEIL_DIV(m, BLOCK_SIZE));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  gemm_kernel<T, BLOCK_SIZE>
      <<<grid, block>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float millisec = 0;
  hipEventElapsedTime(&millisec, start, stop);
  const double sec = double(millisec) / 1e3;
  const double gflops = double(m * n * k * 2) / sec / 1e9;
  [[maybe_unused]] const double band =
      double(m * k + k * n + m * n) * sizeof(T) / sec / 1e9;

  printf("%4ld, %4ld, %4ld, %8.2f, %9.6f\n", m, n, k, gflops, sec);

  CUDA_CHECK(hipMemcpy(C, d_C, m * n * sizeof(T), hipMemcpyDeviceToHost));

  hipEventDestroy(start);
  hipEventDestroy(stop);
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
}

template void gemm_cuda(const size_t m, const size_t n, const size_t k,
                        const float alpha, const float *A, const float *B,
                        const float beta, float *C);
